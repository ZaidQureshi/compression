#include "hip/hip_runtime.h"
#include <common.h>
#include <unistd.h>
#include <iostream>
#include <cstring>
#include <sys/types.h> 
#include <sys/stat.h> 
#include <fcntl.h>
#include <sys/mman.h>

#include <chrono>

#include <rlev2/rlev2.h>

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Please provide arguments\n";
        exit(1);
    }

    int in_fd;
    struct stat in_sb;

    if((in_fd = open(argv[1], O_RDONLY)) == 0) {
        printf("Fatal Error: INPUT File open error\n");
        return -1;
    }
    fstat(in_fd, &in_sb);

    int64_t *in = (int64_t *)mmap(nullptr, in_sb.st_size, PROT_READ, MAP_PRIVATE, in_fd, 0);
    if(in == (void*)-1){
        printf("Fatal Error: INPUT Mapping error\n");
        return -1;
    }
    close(in_fd);

    uint8_t *encoded = nullptr;
    uint64_t encoded_bytes = 0;

    blk_off_t *blk_off;
    col_len_t *col_len;
    uint64_t n_chunks;

    auto encode_start = std::chrono::high_resolution_clock::now();
    rlev2::compress_gpu_transpose(in, in_sb.st_size, encoded, encoded_bytes, n_chunks, blk_off, col_len);
    auto encode_end = std::chrono::high_resolution_clock::now();

    int64_t *decoded = nullptr;
    uint64_t decoded_bytes = 0;

    auto decode_start = std::chrono::high_resolution_clock::now();
    rlev2::decompress_gpu(encoded, encoded_bytes, n_chunks, blk_off, col_len, decoded, decoded_bytes);
    auto decode_end = std::chrono::high_resolution_clock::now();
       
    auto decomp = std::chrono::duration_cast<std::chrono::duration<double>>(decode_end - decode_start);
    std::cout << "Decompression size: " << encoded_bytes << " bytes\n";
    std::cout << "Decompression time: " << decomp.count() << " secs\n";
    // printf("exp(actual) %lu(%lu)\n",decoded_bytes, sizeof(ll));
    // for (int i=0; i<n_digits; ++i) {
    //     if (ll[i] != decoded[i]) {
    //         printf("failed at %d\n", i);
    //         break;

    //     }
    //     // printf("%ld : %ld\n", ll[i], decompressed[i]);
    // }
    
    assert(decoded_bytes == in_sb.st_size);
    for (int i=0; i<decoded_bytes/sizeof(int64_t); ++i) {
         if (decoded[i] != in[i]) {
             printf("fail at %d %ld(%ld)\n", i, in[i], decoded[i]);
         }
        assert(decoded[i] == in[i]);
    }

    delete[] blk_off;
    delete[] col_len;
    delete[] encoded;
    delete[] decoded;
    if(munmap(in, in_sb.st_size) == -1) PRINT_ERROR;
}
