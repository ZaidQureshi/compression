#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdint>

#define MAXBITS 15
#define FIXLCODES 288
#define MaxDCodes 30

struct huffman_tree {

	int16_t lencnt[MAXBITS + 1];
	int16_t lensym[FIXLCODES];
	int16_t distcnt[MAXBITS + 1];
	int16_t distsym[MaxDCodes];

};

__device__
int16_t decode (uint32_t test, const int16_t* counts, const int16_t* symbols){

	unsigned int first;
	unsigned int len;
	unsigned int code;
	unsigned int count;
	uint32_t next32r = __brev(test);


	first  = 0;
  	for (len = 1; len <= MAXBITS; len++) {
    	code  = (next32r >> (32 - len)) - first;
    	
    	count = counts[len];
    if (code < count) 
    {
      	//skipbits(s, len);
      	return symbols[code];
    }
	    symbols += count;  
	    first += count;
	    first <<= 1;
  	}

  return -10;
}

__global__
void d_test_decode_testcase(uint32_t test, const int16_t* counts, const int16_t* symbols){

	int16_t out = decode(test, counts, symbols);
	char c = 'A' + out;
	printf("symbol: %c\n", c);

}

void test_decode_testcase(uint32_t* tests, int num_test, const int16_t* counts, const int16_t* symbols){

	int16_t* d_counts;
	int16_t* d_symbols;

	hipMalloc(&d_counts, (MAXBITS + 1) * sizeof(int16_t));
	hipMalloc(&d_symbols, (FIXLCODES) * sizeof(int16_t));

	hipMemcpy(d_counts, counts, (MAXBITS + 1) * sizeof(int16_t), hipMemcpyHostToDevice);
	hipMemcpy(d_symbols, symbols, (FIXLCODES) * sizeof(int16_t), hipMemcpyHostToDevice);

	for(int i = 0; i < num_test; i++) {
		uint32_t test = tests[i];
		d_test_decode_testcase<<<1,1>>>(test, d_counts, d_symbols);
	}
	
	hipFree(d_counts);
	hipFree(d_symbols);
}

//Construct huffman tree
__device__ 
void construct(int16_t* counts, int16_t* symbols, const int16_t *length, int num_codes){

	int symbol;
	int len;
	int left;
	for(len = 0; len <= MAXBITS; len++)
		counts[len] = 0;

	for(symbol = 0; symbol < num_codes; symbol++)
		(counts[length[symbol]])++;

	left = 1;
	for(len = 1; len <= MAXBITS; len++){
		left <<= 1;
		left -= counts[len];       
    	if (left < 0) 
    		return; 
	}

	{
		//computing offset array for conunts
		int16_t offs[MAXBITS + 1];
		offs[1] = 0;
		for (len = 1; len < MAXBITS; len++)
			offs[len + 1] = offs[len] + counts[len];

		for(symbol = 0; symbol < num_codes; symbol++){
			 if (length[symbol] != 0) 
			 	symbols[offs[length[symbol]]++] = symbol;
		}
	}	
}

__global__ 
void d_test_construct(int16_t* counts, int16_t* symbols, int16_t* length, int num_codes) {

	construct(counts, symbols, length, num_codes);
}

void test_construct_testcase(int16_t* length, int num_codes, int16_t* counts, int16_t* symbols){

	int16_t* d_length;
	hipMalloc(&d_length, num_codes * sizeof(int16_t));
	hipMemcpy(d_length, length, num_codes * sizeof(int16_t), hipMemcpyHostToDevice);

	int16_t* d_counts;
	int16_t* d_symbols;

	hipMalloc(&d_counts, (MAXBITS + 1) * sizeof(int16_t));
	hipMalloc(&d_symbols, (FIXLCODES) * sizeof(int16_t));

	printf("test construct kernel launch\n");
	d_test_construct<<<1,1>>>(d_counts, d_symbols, d_length, num_codes);

	hipMemcpy(counts, d_counts, (MAXBITS + 1) * sizeof(int16_t), hipMemcpyDeviceToHost);
	hipMemcpy(symbols, d_symbols, (FIXLCODES) * sizeof(int16_t), hipMemcpyDeviceToHost);

	hipFree(d_counts);
	hipFree(d_symbols);
}



void test_construct(){

	int16_t counts [MAXBITS + 1];
	int16_t symbols [FIXLCODES];



	printf("test1 starts\n");
	//test 1
	int num_codes = 4;
	int16_t length[4] = {2, 1, 3, 3};
	uint32_t tests1[4] = {1, 0, 3, 7};
	test_construct_testcase(length, num_codes, counts, symbols);

	for(int i = 0; i < 10; i++){
		printf("counts: %i \t symbols: %i \n", counts[i], symbols[i]);
	}

	test_decode_testcase(tests1, 4, counts, symbols);

	printf("test2 starts\n");
	//test 2
	num_codes = 8;
	int16_t length2[8] = {3, 3, 3, 3, 3, 2, 4, 4};
	test_construct_testcase(length2, num_codes, counts, symbols);
	
	for(int i = 0; i < 10; i++){
		printf("counts: %i \t symbols: %i \n", counts[i], symbols[i]);
	}

}



//construct huffman tree for dynamic huffman encoding block
__device__
int decode_dynamic(){
	int hlit, hdist, hclen;

	//getting the meta data for the compressed block
	fetch_n_bits<int>(5, hlit);
	fetch_n_bits<int>(5, hdist);
	fetch_n_bits<int>(4, hclen);
	hlit += 257;
	hdist += 1;
	hclen += 4;
 
 	//check
 	for (index = 0; index < ncode; index++) lengths[g_code_order[index]] = getbits(s, 3);
  	for (; index < 19; index++) lengths[g_code_order[index]] = 0;


	construct(s, s->lencnt, s->lensym, lengths, 19);

	
	int index = 0;
	while (index < nlen + ndist) {
		int symbol = decode(s, s->lencnt, s->lensym);
		if (symbol < 0) return symbol; //error

		//represent code lengths of 0 - 15
		if(symbol < 16)
			lengths[index++] = symbol;

		else{
			int len = 0;
			if(symbol == 16) {
				 len = lengths[index - 1];  // last length
				 fetch_n_bits<int>(2, symbol);
				 symbol += 3;
			}
			else if(symbol == 17){
				fetch_n_bits<int>(3, symbol);
				symbol += 3;
			}
			else {
				fetch_n_bits<int>(7, symbol);
				symbol += 11;
			}

			while(symbol--)
				lenghts[index++] = len;
		}
	}

	//check
	if(lengts[256] == 0) return -9;


	construct(s, s->lencnt, s->lensym, lengths, nlen);
    
    construct(s, s->distcnt, s->distsym, &lengths[nlen], ndist);

	return 0;
}


//code starts from 257
static const __device__ __constant__ uint16_t g_lens[29] = {  // Size base for length codes 257..285
  3,  4,  5,  6,  7,  8,  9,  10, 11,  13,  15,  17,  19,  23, 27,
  31, 35, 43, 51, 59, 67, 83, 99, 115, 131, 163, 195, 227, 258};

//code starts from 257
static const __device__ __constant__ uint16_t
  g_lext[29] = { 
    0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3, 4, 4, 4, 4, 5, 5, 5, 5, 0};


static const __device__ __constant__ uint16_t
  g_dists[30] = {  // Offset base for distance codes 0..29
    1,   2,   3,   4,   5,   7,    9,    13,   17,   25,   33,   49,   65,    97,    129,
    193, 257, 385, 513, 769, 1025, 1537, 2049, 3073, 4097, 6145, 8193, 12289, 16385, 24577};

static const __device__ __constant__ uint16_t g_dext[30] = {  // Extra bits for distance codes 0..29
  0, 0, 0, 0, 1, 1, 2, 2, 3, 3, 4, 4, 5, 5, 6, 6, 7, 7, 8, 8, 9, 9, 10, 10, 11, 11, 12, 12, 13, 13};



//decode code for compressed block
__device__ decode_symbol() {


	uint16_t sym = decode(test,  s->lencnt, s->lensym);


	//parse 5 bits

	//not compressed, literal
	if(sym < 255) {

	}

	//end of block
	else if(sym == 256) {

	}

	//lenght, need to parse
	else{
		uint16_t extra_bits = g_lext[sym - 257];

		uint16_t extra_len  = 0;
		if(extra_bits != 0)
			fetch_n_bits<uint16_t>(extra_bits, extra_len);

		uint16_t len = extra_len + g_lens[sym - 257];


		//distance, 5bits
		uint16_t sym_dist = decode(test,  s->distcnt, s->distsym);
		uint16_t extra_bits_dist = g_dext[sym_dist];
		
		uint16_t extra_len_dist = 0;
		if(extra_bits_dist != 0)
			fetch_n_bits<uint16_t>(extra_bits_dist, extra_len_dist);

		uint16_t dist = extra_len_dist + g_dists[dist];

		//(len, dist_len)

	}

}



int main(int argc, char** argv) {

	test_construct();
	return 0;

}